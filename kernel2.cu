#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of 
 * students taking the CSC367 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited. 
 * All forms of distribution of this code, whether as given or with 
 * any changes, are expressly prohibited. 
 * 
 * Authors: Bogdan Simion, Maryam Dehnavi, Felipe de Azevedo Piovezan
 * 
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2020 Bogdan Simion and Maryam Dehnavi
 * -------------
*/

#include "kernels.h"
#include <stdio.h>
#include <string>
#include <unistd.h>
#include <sys/sysinfo.h>
#include <time.h>
#include <stdint.h>
#define max_threads 1024

void run_kernel2(const int8_t *filter, int32_t dimension, const int32_t *input,
                  int32_t *output, int32_t width, int32_t height)
{
    // Calculate blocks and threads
    int pixel_count = width * height;

    int32_t num_threads = min(max_threads, pixel_count);
    int32_t num_blocks = (pixel_count + num_threads - 1) / num_threads;

    int32_t blocks_reduction = (pixel_count + num_threads - 1) / num_threads;
    // printf("num_threads: %ld, num_blocks: %ld \n", num_threads, num_blocks);

    // kernel 2
    dim3 dimBlock(num_threads, 1, 1);
  	dim3 dimGrid(num_blocks, 1, 1);
  	kernel2<<<dimGrid, dimBlock>>>(filter, dimension, input, output, width, height);

    // init global min & max 
    int32_t *global_min;
    int32_t *global_max;
    hipMalloc(&global_min, width*height*sizeof(int32_t));
    hipMalloc(&global_max, width*height*sizeof(int32_t));
    hipMemcpy(global_min, output, width * height * sizeof(int32_t), hipMemcpyDeviceToDevice);
    hipMemcpy(global_max, output, width * height * sizeof(int32_t), hipMemcpyDeviceToDevice);

    int shMemSize = (num_threads <= 32) ? 4 * num_threads * sizeof(int32_t) : 2* num_threads * sizeof(int32_t);
    reduction<<<blocks_reduction, num_threads, shMemSize>>>(global_min, global_max, pixel_count);

    while (blocks_reduction > 1) {
        int n = blocks_reduction;
        blocks_reduction = (blocks_reduction + max_threads - 1) / max_threads;

        shMemSize = (num_threads <= 32) ? 4 * num_threads * sizeof(int32_t) : 2* num_threads * sizeof(int32_t);
        reduction<<<blocks_reduction, num_threads, shMemSize>>>(global_min, global_max, n);
    }

    // normalize 2
    normalize2<<<num_blocks, num_threads>>>(output, width, height, global_min, global_max);
    hipFree(global_min);
    hipFree(global_max);
}


/*row major*/
__global__ void kernel2(const int8_t *filter, int32_t dimension, 
        const int32_t *input, int32_t *output, int32_t width, int32_t height)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row = idx / width;
	int column = idx % width;

    if (idx < width * height){
        output[idx] = apply2d_gpu(filter, dimension, input, width, height, row, column);
    }
}


__global__ void normalize2(int32_t *image, int32_t width, int32_t height,
        int32_t *smallest, int32_t *biggest)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

  	if (idx < height * width){
    	if (smallest[0] != biggest[0]){
      		image[idx] = ((image[idx] - smallest[0]) * 255) / (biggest[0] - smallest[0]);
    }
  }
}
